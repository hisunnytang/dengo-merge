#include "hip/hip_runtime.h"
// includes, system
#include <cstdlib>
#include <math.h>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <stdio.h>
// includes, cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

texture<float, 1, hipReadModeElementType> data_d_texture_filtering;
texture<float, 1> data_d_texture;

#define BLOCK_SIZE 256

/******************/
/* ERROR CHECKING */
/******************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__);  }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code);  }

    }

}
/************/
/* LINSPACE */
/************/
// --- Generates N equally spaced, increasing points between a and b and stores them in x
void linspace(float* x, float a, float b, int N) {
    float delta_x=(b-a)/(float)N;
    x[0]=a;
    for(int k=1;k<N;k++) x[k]=x[k-1]+delta_x;

}

/*************/
/* RANDSPACE */
/*************/
// --- Generates N randomly spaced, increasing points between a and b and stores them in x
void randspace(float* x, float a, float b, int N) {
    float delta_x=(b-a)/(float)N;
    x[0]=a;
    for(int k=1;k<N;k++) x[k]=x[k-1]+delta_x+(((float)rand()/(float)RAND_MAX-0.5)*(1./(float)N));

}

/******************/
/* DATA GENERATOR */
/******************/
// --- Generates N complex random data points, with real and imaginary parts ranging in (0.f,1.f)
void Data_Generator(float* data, int N) {
    for(int k=0;k<N;k++) {
        data[k]=(float)rand()/(float)RAND_MAX;

    }

}

/*************************************/
/* LINEAR INTERPOLATION KERNEL - CPU */
/*************************************/
float linear_kernel_CPU(float in)
{
    float d_y;
    return 1.-abs(in);

}

/***************************************/
/* LINEAR INTERPOLATION FUNCTION - CPU */
/***************************************/
void linear_interpolation_function_CPU(float* result_GPU, float* data, float* x_in, float* x_out, int M, int N){

    float a;
    for(int j=0; j<N; j++){
        int k = floor(x_out[j]+M/2);
        a = x_out[j]+M/2-floor(x_out[j]+M/2);
        result_GPU[j] = a * data[k+1] + (-data[k] * a + data[k]);

    }


}

/*************************************/
/* LINEAR INTERPOLATION KERNEL - GPU */
/*************************************/
__device__ float linear_kernel_GPU(float in)
{
    float d_y;
    return 1.-abs(in);

}

/**************************************************************/
/* LINEAR INTERPOLATION KERNEL FUNCTION - GPU - GLOBAL MEMORY */
/**************************************************************/
__global__ void linear_interpolation_kernel_function_GPU(float* __restrict__ result_d, const float* __restrict__ data_d, const float* __restrict__ x_out_d, const int M, const int N)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if(j<N)
    {
        float reg_x_out = x_out_d[j]+M/2;
        int k = __float2int_rz(reg_x_out);
        float a = reg_x_out - truncf(reg_x_out);
        float dk = data_d[k];
        float dkp1 = data_d[k+1];
        result_d[j] = a * dkp1 + (-dk * a + dk);

    }

}

/***************************************************************/
/* LINEAR INTERPOLATION KERNEL FUNCTION - GPU - TEXTURE MEMORY */
/***************************************************************/
__global__ void linear_interpolation_kernel_function_GPU_texture(float* __restrict__ result_d, const float* __restrict__ x_out_d, const int M, const int N)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if(j<N)
    {
        float reg_x_out = x_out_d[j]+M/2;
        int k = __float2int_rz(reg_x_out);
        float a = reg_x_out - truncf(reg_x_out);
        float dk = tex1Dfetch(data_d_texture,k);
        float dkp1 = tex1Dfetch(data_d_texture,k+1);
        result_d[j] = a * dkp1 + (-dk * a + dk);

    }

}

/************************************************************************************/
/* LINEAR INTERPOLATION KERNEL FUNCTION - GPU - TEXTURE MEMORY - FILTERING FEATURES */
/************************************************************************************/
__global__ void linear_interpolation_kernel_function_GPU_texture_filtering(float* __restrict__ result_d, const float* __restrict__ x_out_d, const int M, const int N)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    if(j<N) result_d[j] = tex1D(data_d_texture_filtering,float(x_out_d[j]+M/2+0.5));

}

/***************************************/
/* LINEAR INTERPOLATION FUNCTION - GPU */
/***************************************/
void linear_interpolation_function_GPU(float* result_d, float* data_d, float* x_in_d, float* x_out_d, int M, int N){

    dim3 dimBlock(BLOCK_SIZE,1); dim3 dimGrid(N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1),1);
    linear_interpolation_kernel_function_GPU<<<dimGrid,dimBlock>>>(result_d, data_d, x_out_d, M, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

}

/********************************************************/
/* LINEAR INTERPOLATION FUNCTION - GPU - TEXTURE MEMORY */
/********************************************************/
void linear_interpolation_function_GPU_texture(float* result_d, float* data_d, float* x_in_d, float* x_out_d, int M, int N){

    hipBindTexture(NULL, data_d_texture, data_d, M*sizeof(float));

    dim3 dimBlock(BLOCK_SIZE,1); dim3 dimGrid(N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1),1);
    linear_interpolation_kernel_function_GPU_texture<<<dimGrid,dimBlock>>>(result_d, x_out_d, M, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

}

/*****************************************************************************/
/* LINEAR INTERPOLATION FUNCTION - GPU - TEXTURE MEMORY - FILTERING FEATURES */
/*****************************************************************************/
void linear_interpolation_function_GPU_texture_filtering(float* result_d, float* data, float* x_in_d, float* x_out_d, int M, int N){

    hipArray* data_d = NULL; gpuErrchk(hipMallocArray(&data_d, &data_d_texture_filtering.channelDesc, M, 1));
    gpuErrchk(hipMemcpyToArray(data_d, 0, 0, data, sizeof(float)*M, hipMemcpyHostToDevice));
    gpuErrchk(hipBindTextureToArray(data_d_texture_filtering, data_d));
    data_d_texture_filtering.normalized = false;
    data_d_texture_filtering.filterMode = hipFilterModeLinear;

    dim3 dimBlock(BLOCK_SIZE,1); dim3 dimGrid(N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0:1),1);
    linear_interpolation_kernel_function_GPU_texture_filtering<<<dimGrid,dimBlock>>>(result_d, x_out_d, M, N);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());


}
/********/
/* MAIN */
/********/
int main()
{

    int M=1024;                // --- Number of input points

    int N=1024;                // --- Number of output points

    int Nit = 1000;            // --- Number of computations for time measurement

    // --- Input sampling
    float* x_in; gpuErrchk(hipMallocManaged(&x_in,sizeof(float)*M));

    // --- Input data
    float *data;        gpuErrchk(hipMallocManaged(&data,(M+1)*sizeof(float))); Data_Generator(data,M); data[M]=0.;

    // --- Output sampling
    float* x_out;        gpuErrchk(hipMallocManaged((void**)&x_out,sizeof(float)*N)); randspace(x_out,-M/2.,M/2.,N);

    // --- Result allocation
    float *result_CPU;                            result_CPU=(float*)malloc(N*sizeof(float));
    float *result_d;                            gpuErrchk(hipMallocManaged(&result_d,sizeof(float)*N));
    float *result_d_texture;                    gpuErrchk(hipMallocManaged(&result_d_texture,sizeof(float)*N));
    float *result_d_texture_filtering;            gpuErrchk(hipMallocManaged(&result_d_texture_filtering,sizeof(float)*N));

    // --- Reference interpolation result as evaluated on the CPU
    linear_interpolation_function_CPU(result_CPU, data, x_in, x_out, M, N);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int k=0; k<Nit; k++) linear_interpolation_function_GPU(result_d, data, x_in, x_out, M, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout << "GPU Global memory [ms]: " << setprecision (10) << time/Nit << endl;

    hipEventRecord(start, 0);
    for (int k=0; k<Nit; k++) linear_interpolation_function_GPU_texture_filtering(result_d_texture_filtering, data, x_in, x_out, M, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout << "GPU Texture filtering [ms]: " << setprecision (10) << time/Nit << endl;

    hipEventRecord(start, 0);
    for (int k=0; k<Nit; k++) linear_interpolation_function_GPU_texture(result_d_texture, data, x_in, x_out, M, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout << "GPU Texture [ms]: " << setprecision (10) << time/Nit << endl;

    float diff_norm=0.f, norm=0.f;
    for(int j=0; j<N; j++) {
        diff_norm = diff_norm + (result_CPU[j]-result_d[j])*(result_CPU[j]-result_d[j]);
        norm      = norm      + result_CPU[j]*result_CPU[j];

    }
    printf("Error GPU [percentage] = %f\n",100.*sqrt(diff_norm/norm));

    float diff_norm_texture_filtering=0.f;
    for(int j=0; j<N; j++) {
        diff_norm_texture_filtering = diff_norm_texture_filtering + (result_CPU[j]-result_d_texture_filtering[j])*(result_CPU[j]-result_d_texture_filtering[j]);

    }
    printf("Error texture filtering [percentage] = %f\n",100.*sqrt(diff_norm_texture_filtering/norm));

    float diff_norm_texture=0.f;
    for(int j=0; j<N; j++) {
        diff_norm_texture = diff_norm_texture + (result_CPU[j]-result_d_texture[j])*(result_CPU[j]-result_d_texture[j]);

    }
    printf("Error texture [percentage] = %f\n",100.*sqrt(diff_norm_texture/norm));

    hipDeviceReset();

    return 0;

}
