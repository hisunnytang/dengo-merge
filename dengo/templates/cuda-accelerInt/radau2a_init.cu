#include "hip/hip_runtime.h"
/**
 * \file
 * \brief Implementation of the necessary initialization for the Radau-IIA solver
 *
 * \author Nicholas Curtis
 * \date 03/09/2015
 *
 */

 #include "solver_init.cuh"

#ifdef GENERATE_DOCS
namespace radau2acu {
#endif

 void init_solver_log() {

 }

 void solver_log() {

 }

/*!
   \fn char* solver_name()
   \brief Returns a descriptive solver name
*/
 const char* solver_name() {
 	const char* name = "radau2a-int-gpu";
 	return name;
 }

 /*!
   \brief Returns the total size (in bytes) required for memory storage for a single GPU thread
   Used in calculation of the maximum number of possible GPU threads to launch, this method
   returns the size of the solver_memory structure (per-GPU thread)
   @see solver_memory
 */
 size_t required_solver_size() {
 	//return the size (in bytes), needed per cuda thread
 	size_t num_bytes = 0;
  //regular jacobian factorization
  num_bytes += NSP * NSP * sizeof(double);
  //complex jacobian factorization
  num_bytes += NSP * NSP * sizeof(hipDoubleComplex);
 	//an error scale array
 	num_bytes += NSP * sizeof(double);
  //two pivot index arrays
  num_bytes += 2 * NSP * sizeof(int);
 	//6 RHS and interpolant arrays
 	num_bytes += 6 * NSP * sizeof(double);
 	//continuation array of size 3 * NSP
 	num_bytes += 3 * NSP * sizeof(double);
 	//y0
 	num_bytes += NSP * sizeof(double);
 	//3 work arrays
 	num_bytes += 3 * NSP * sizeof(double);
  //1 complex work array
  num_bytes += NSP * sizeof(double);
  //result flag
  num_bytes += 1 * sizeof(int);

  return num_bytes;
 }

/*!
 * \brief Convienvience method to Cuda Malloc and memset a pointer to zero
 * \param ptr The address of the pointer to malloc
 * \param size The total size (in bytes) of the pointer to malloc
 */
void createAndZero(void** ptr, size_t size)
{
  cudaErrorCheck(hipMalloc(ptr, size));
  cudaErrorCheck(hipMemset(*ptr, 0, size));
}

/*!
   \brief Solves for the poles and residuals used for the Rational Approximants in the Krylov subspace methods and initializes solver_memory
   \param padded The total (padded) number of GPU threads (IVPs) to solve
   \param h_mem The host solver_memory structure (to be copied to the GPU)
   \param d_mem The device solver_memory structure (to be operated on by the GPU)
*/
void initialize_solver(const int padded, solver_memory** h_mem, solver_memory** d_mem) {
  // Allocate storage for the device struct
  cudaErrorCheck( hipMalloc(d_mem, sizeof(solver_memory)) );
  //allocate the device arrays on the host pointer
  createAndZero((void**)&((*h_mem)->E1), NSP * NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->E2), NSP * NSP * padded * sizeof(hipDoubleComplex));
  createAndZero((void**)&((*h_mem)->scale), NSP * padded * sizeof(hipDoubleComplex));
  createAndZero((void**)&((*h_mem)->ipiv1), NSP * padded * sizeof(int));
  createAndZero((void**)&((*h_mem)->ipiv2), NSP * padded * sizeof(int));
  createAndZero((void**)&((*h_mem)->Z1), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->Z2), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->Z3), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->DZ1), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->DZ2), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->DZ3), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->CONT), 3 * NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->y0), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->work1), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->work2), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->work3), NSP * padded * sizeof(double));
  createAndZero((void**)&((*h_mem)->work4), NSP * padded * sizeof(hipDoubleComplex));
  createAndZero((void**)&((*h_mem)->result), padded * sizeof(int));

  //copy host struct to device
  cudaErrorCheck( hipMemcpy(*d_mem, *h_mem, sizeof(solver_memory), hipMemcpyHostToDevice) );
}

/*!
   \fn void cleanup_solver(solver_memory** h_mem, solver_memory** d_mem)
   \brief Cleans up solver memory, and closes Krylov subspace logfiles (if LOG_OUTPUT is defined)
   @see solver_memory
   @see solver_options.cuh
*/
 void cleanup_solver(solver_memory** h_mem, solver_memory** d_mem) {
  cudaErrorCheck(hipFree((*h_mem)->E1));
  cudaErrorCheck(hipFree((*h_mem)->E2));
  cudaErrorCheck(hipFree((*h_mem)->scale));
  cudaErrorCheck(hipFree((*h_mem)->ipiv1));
  cudaErrorCheck(hipFree((*h_mem)->ipiv2));
  cudaErrorCheck(hipFree((*h_mem)->Z1));
  cudaErrorCheck(hipFree((*h_mem)->Z2));
  cudaErrorCheck(hipFree((*h_mem)->Z3));
  cudaErrorCheck(hipFree((*h_mem)->DZ1));
  cudaErrorCheck(hipFree((*h_mem)->DZ2));
  cudaErrorCheck(hipFree((*h_mem)->DZ3));
  cudaErrorCheck(hipFree((*h_mem)->CONT));
  cudaErrorCheck(hipFree((*h_mem)->y0));
  cudaErrorCheck(hipFree((*h_mem)->work1));
  cudaErrorCheck(hipFree((*h_mem)->work2));
  cudaErrorCheck(hipFree((*h_mem)->work3));
  cudaErrorCheck(hipFree((*h_mem)->work4));
  cudaErrorCheck(hipFree((*h_mem)->result));
  cudaErrorCheck(hipFree(*d_mem));
}
