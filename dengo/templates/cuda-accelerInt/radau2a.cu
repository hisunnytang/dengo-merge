#include "hip/hip_runtime.h"
/**
* \file
*
* \author Nicholas J. Curtis
* \date 03/16/2015
*
* \brief A Radau2A IRK implementation for CUDA
* Adapted from Hairer and Wanner's [RADAU5 code](http://www.unige.ch/~hairer/prog/stiff/radau5.f)
* and the [FATODE](http://people.cs.vt.edu/~asandu/Software/FATODE/index.html) ODE integration library
*
* For full reference see:\n
* G. Wanner, E. Hairer, Solving Ordinary Differential Equations II: Stiff and DifferentialAlgebraic
Problems, 2nd Edition, Springer-Verlag, Berlin, 1996. doi:10.1007/978-3-642-
05221-7.
*
* NOTE: all matricies stored in column major format!
*
*/

#include <hip/hip_complex.h>

//various mechanism/solver defns
//these should be included first
#include "header.cuh"
#include "solver_options.cuh"
#include "solver_props.cuh"

//math operations
#include "inverse.cuh"
#include "complexInverse.cuh"

//rate/jacobian subroutines
#ifndef FINITE_DIFFERENCE
#include "jacob.cuh"
#else
#include "fd_jacob.cuh"
#endif
#include "dydt.cuh"
#include "gpu_macros.cuh"

#ifdef GENERATE_DOCS
namespace radau2acu {
#endif

//#define WARP_VOTING
#ifdef WARP_VOTING
	#define ANY(X) (__any((X)))
	#define ALL(X) (__all((X)))
#else
	#define ANY(X) ((X))
	#define ALL(X) ((X))
#endif

//! Maximum number of allowed internal timesteps before error
#define Max_no_steps (2000000)
//! Maximum number of allowed Newton iteration steps before error
#define NewtonMaxit (8)
//! Use quadratic interpolation from previous step if possible
#define StartNewton (true)
//! Use gustafsson time stepping control
#define Gustafsson
//! Smallist representable double precision number
#define Roundoff (EPS)
//! Controls maximum decrease in timestep size
#define FacMin (0.2)
//! Controls maximum increase in timestep size
#define FacMax (8)
//! Safety factor for Gustafsson time stepping control
#define FacSafe (0.9)
//! Time step factor on rejected step
#define FacRej (0.1)
//! Minimum Newton convergence rate
#define ThetaMin (0.001)
//! Newton convergence tolerance
#define NewtonTol (0.03)
//! Min Timestep ratio to skip LU decomposition
#define Qmin (1.0)
//! Max Timestep ratio to skip LU decomposition
#define Qmax (1.2)
//#define UNROLL (8)
//! Error allowed on this many consecutive internal timesteps before exit
#define Max_consecutive_errs (5)
//#define UNROLL (8)
#ifdef DIVERGENCE_TEST
 	extern __device__ int integrator_steps[DIVERGENCE_TEST];
#endif
//#define SDIRK_ERROR

/**
 * \brief Computes error weight scaling from initial and current state
 * \param[in]		y0			the initial state vector to use
 * \param[in]		y			the current state vector
 * \param[out]		sc			the populated error weight scalings
 */
__device__
void scale (double const * const __restrict__ y0,
			double const * const __restrict__ y,
			double * const __restrict__ sc) {
	#pragma unroll 8
	for (int i = 0; i < NSP; ++i) {
		sc[INDEX(i)] = 1.0 / (ATOL + fmax(fabs(y0[INDEX(i)]), fabs(y[INDEX(i)])) * RTOL);
	}
}

/**
 * \brief Computes error weight scaling from initial state
 * \param[in]		y0			the initial state vector to use
 * \param[out]		sc			the populated error weight scalings
 */
__device__
void scale_init (double const * const __restrict__ y0,
				 double * const __restrict__ sc) {
	#pragma unroll 8
	for (int i = 0; i < NSP; ++i) {
		sc[INDEX(i)] = 1.0 / (ATOL + fabs(y0[INDEX(i)]) * RTOL);
	}
}

/**
 * \brief A convienence method that provides an unrolled memcpy for CUDA code
 * \param[out]		dest			The destination vector
 * \param[in]		source			The source vector
 */
__device__
void safe_memcpy(double * const __restrict__ dest,
				 double const * const __restrict__ source)
{
	#pragma unroll 8
	for (int i = 0; i < NSP; i++)
	{
		dest[INDEX(i)] = source[INDEX(i)];
	}
}

/**
 * \brief A convienence method that provides an unrolled memset three vectors in CUDA code
 * \param[out]		dest1			The first destination vector
 * \param[out]		dest2			The second destination vector
 * \param[out]		dest3			The third destination vector
 * \param[in]		val				The value to use in memset
 */
__device__
void safe_memset3(double * const __restrict__ dest1,
				  double * const __restrict__ dest2,
				  double * const __restrict__ dest3, const double val)
{
	#pragma unroll 8
	for (int i = 0; i < NSP; i++)
	{
		dest1[INDEX(i)] = val;
		dest2[INDEX(i)] = val;
		dest3[INDEX(i)] = val;
	}
}

/**
 * \brief A convienence method that provides an unrolled memset of a single vector in CUDA code
 * \param[out]		dest			The destination vector
 * \param[in]		val				The value to use in memset
 */
__device__
void safe_memset(double * const __restrict__ dest1, const double val)
{
	#pragma unroll 8
	for (int i = 0; i < NSP; i++)
	{
		dest1[INDEX(i)] = val;
	}
}

/**
 * \brief A convienence method that provides an unrolled memset of 2D (NSP x NSP) matrix in CUDA code
 * \param[out]		dest1			The destination vector
 * \param[in]		val				The value to use in memset
 */
__device__
void safe_memset_jac(double * const __restrict__ dest1, const double val)
{
	#pragma unroll 8
	for (int i = 0; i < NSP * NSP; i++)
	{
		dest1[INDEX(i)] = val;
	}
}

/**
 * \defgroup RK_Params Various parameters for the RadauIIA method
 * @{
 */

__constant__ double rkA[3][3] = { {
	 1.968154772236604258683861429918299e-1,
	-6.55354258501983881085227825696087e-2,
	 2.377097434822015242040823210718965e-2
	}, {
	 3.944243147390872769974116714584975e-1,
	 2.920734116652284630205027458970589e-1,
	-4.154875212599793019818600988496743e-2
	}, {
	 3.764030627004672750500754423692808e-1,
	 5.124858261884216138388134465196080e-1,
	 1.111111111111111111111111111111111e-1
	}
};

__constant__ double rkB[3] = {
3.764030627004672750500754423692808e-1,
5.124858261884216138388134465196080e-1,
1.111111111111111111111111111111111e-1
};

__constant__ double rkC[3] = {
1.550510257216821901802715925294109e-1,
6.449489742783178098197284074705891e-1,
1.0
};

//Local order of error estimator
/*
!~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
!~~~> Diagonalize the RK matrix:
! rkTinv * inv(rkA) * rkT =
!           |  rkGamma      0           0     |
!           |      0      rkAlpha   -rkBeta   |
!           |      0      rkBeta     rkAlpha  |
!~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
*/

__constant__ double rkGamma = 3.637834252744495732208418513577775e0;
__constant__ double rkAlpha = 2.681082873627752133895790743211112e0;
__constant__ double rkBeta  = 3.050430199247410569426377624787569e0;

__constant__ double rkT[3][3] = {
{9.443876248897524148749007950641664e-2,
-1.412552950209542084279903838077973e-1,
-3.00291941051474244918611170890539e-2},
{2.502131229653333113765090675125018e-1,
2.041293522937999319959908102983381e-1,
3.829421127572619377954382335998733e-1},
{1.0e0,
1.0e0,
0.0e0}
};

__constant__ double rkTinv[3][3] = {
{4.178718591551904727346462658512057e0,
3.27682820761062387082533272429617e-1,
5.233764454994495480399309159089876e-1},
{-4.178718591551904727346462658512057e0,
-3.27682820761062387082533272429617e-1,
4.766235545005504519600690840910124e-1},
{-5.02872634945786875951247343139544e-1,
2.571926949855605429186785353601676e0,
-5.960392048282249249688219110993024e-1}
};

__constant__ double rkTinvAinv[3][3] = {
{1.520148562492775501049204957366528e+1,
1.192055789400527921212348994770778e0,
1.903956760517560343018332287285119e0},
{-9.669512977505946748632625374449567e0,
-8.724028436822336183071773193986487e0,
3.096043239482439656981667712714881e0},
{-1.409513259499574544876303981551774e+1,
5.895975725255405108079130152868952e0,
-1.441236197545344702389881889085515e-1}
};

__constant__ double rkAinvT[3][3] = {
{0.3435525649691961614912493915818282e0,
-0.4703191128473198422370558694426832e0,
0.3503786597113668965366406634269080e0},
{0.9102338692094599309122768354288852e0,
1.715425895757991796035292755937326e0,
0.4040171993145015239277111187301784e0},
{3.637834252744495732208418513577775e0,
2.681082873627752133895790743211112e0,
-3.050430199247410569426377624787569e0}
};

// Classical error estimator:
// H* Sum (B_j-Bhat_j)*f(Z_j) = H*E(0)*f(0) + Sum E_j*Z_j
__constant__ double rkE[4] = {
0.05,
-10.04880939982741556246032950764708*0.05,
1.382142733160748895793662840980412*0.05,
-0.3333333333333333333333333333333333*0.05
};
/*
// H* Sum Bgam_j*f(Z_j) = H*Bgam(0)*f(0) + Sum Theta_j*Z_j
const static double rkTheta[3] = {
-1.520677486405081647234271944611547 - 10.04880939982741556246032950764708*0.05,
2.070455145596436382729929151810376 + 1.382142733160748895793662840980413*0.05,
-0.3333333333333333333333333333333333*0.05 - 0.3744441479783868387391430179970741
};*/

__constant__ double rkELO = 4;

/**
 @}
 */


///////////////////////////////////////////////////////////////////////////////

/*
* calculate E1 & E2 matricies and their LU Decomposition
*/
__device__ void RK_Decomp(double H, const double* const __restrict__ Jac,
							const solver_memory* const __restrict__ solver,
							int* __restrict__ info) {
	double* const __restrict__ E1 = solver->E1;
	hipDoubleComplex* const __restrict__ E2 = solver->E2;
	int* const __restrict__ ipiv1 = solver->ipiv1;
	int* const __restrict__ ipiv2 = solver->ipiv2;
	hipDoubleComplex temp = make_hipDoubleComplex(rkAlpha/H, rkBeta/H);
	#pragma unroll 8
	for (int i = 0; i < NSP; i++)
	{
		#pragma unroll 8
		for(int j = 0; j < NSP; j++)
		{
			E1[INDEX(i + j * NSP)] = -Jac[INDEX(i + j * NSP)];
			E2[INDEX(i + j * NSP)] = make_hipDoubleComplex(-Jac[INDEX(i + j * NSP)], 0);
		}
		E1[INDEX(i + i * NSP)] += rkGamma / H;
		E2[INDEX(i + i * NSP)] = hipCadd(E2[INDEX(i + i * NSP)], temp);
	}
	getLU(NSP, E1, ipiv1, info);
	if (*info != 0) {
		return;
	}
	getComplexLU(NSP, E2, ipiv2, info);
}

__device__ void RK_Make_Interpolate(const double* __restrict__ Z1, const double* __restrict__ Z2,
										const double* __restrict__ Z3, double* __restrict__ CONT) {
	double den = (rkC[2] - rkC[1]) * (rkC[1] - rkC[0]) * (rkC[0] - rkC[2]);
	#pragma unroll 8
	for (int i = 0; i < NSP; i++) {
		CONT[INDEX(i)] = ((-rkC[2] * rkC[2] * rkC[1] * Z1[INDEX(i)] + Z3[INDEX(i)] * rkC[1]* rkC[0] * rkC[0]
                    + rkC[1] * rkC[1] * rkC[2] * Z1[INDEX(i)] - rkC[1] * rkC[1] * rkC[0] * Z3[INDEX(i)]
                    + rkC[2] * rkC[2] * rkC[0] * Z2[INDEX(i)] - Z2[INDEX(i)] * rkC[2] * rkC[0] * rkC[0])
                    /den) - Z3[INDEX(i)];
        CONT[INDEX(NSP + i)] = -( rkC[0] * rkC[0] * (Z3[INDEX(i)] - Z2[INDEX(i)]) + rkC[1] * rkC[1] * (Z1[INDEX(i)] - Z3[INDEX(i)])
        				 + rkC[2] * rkC[2] * (Z2[INDEX(i)] - Z1[INDEX(i)]) )/den;
        CONT[INDEX(NSP + NSP + i)] = ( rkC[0] * (Z3[INDEX(i)] - Z2[INDEX(i)]) + rkC[1] * (Z1[INDEX(i)] - Z3[INDEX(i)])
                           + rkC[2] * (Z2[INDEX(i)] - Z1[INDEX(i)]) ) / den;
	}
}

__device__ void RK_Interpolate(double H, double Hold, double* __restrict__ Z1,
								double* __restrict__ Z2, double* __restrict__ Z3, const double* __restrict__ CONT) {
	double r = H / Hold;
	register double x1 = 1.0 + rkC[0] * r;
	register double x2 = 1.0 + rkC[1] * r;
	register double x3 = 1.0 + rkC[2] * r;
	#pragma unroll 8
	for (int i = 0; i < NSP; i++) {
		Z1[INDEX(i)] = CONT[INDEX(i)] + x1 * (CONT[INDEX(NSP + i)] + x1 * CONT[INDEX(NSP + NSP + i)]);
		Z2[INDEX(i)] = CONT[INDEX(i)] + x2 * (CONT[INDEX(NSP + i)] + x2 * CONT[INDEX(NSP + NSP + i)]);
		Z3[INDEX(i)] = CONT[INDEX(i)] + x2 * (CONT[INDEX(NSP + i)] + x3 * CONT[INDEX(NSP + NSP + i)]);
	}
}


__device__ void WADD(const double* __restrict__ X, const double* __restrict__ Y, double* __restrict__ Z) {
	#pragma unroll 8
	for (int i = 0; i < NSP; i++)
	{
		Z[INDEX(i)] = X[INDEX(i)] + Y[INDEX(i)];
	}
}

__device__ void DAXPY3(double DA1, double DA2, double DA3,
						const double* __restrict__ DX, double* __restrict__ DY1,
						double* __restrict__ DY2, double* __restrict__ DY3) {
	#pragma unroll 8
	for (int i = 0; i < NSP; i++) {
		DY1[INDEX(i)] += DA1 * DX[INDEX(i)];
		DY2[INDEX(i)] += DA2 * DX[INDEX(i)];
		DY3[INDEX(i)] += DA3 * DX[INDEX(i)];
	}
}

/*
*Prepare the right-hand side for Newton iterations
*     R = Z - hA * F
*/
__device__ void RK_PrepareRHS(double t, double pr, double H,
								double const * const __restrict__ Y,
								const solver_memory* __restrict__ solver,
								const mechanism_memory* __restrict__ mech,
								double* __restrict__ TMP,
								double* __restrict__ F) {
	double const * const __restrict__ Z1 = solver->Z1;
	double const * const __restrict__ Z2 = solver->Z2;
	double const * const __restrict__ Z3 = solver->Z3;
	double * const __restrict__ R1 = solver->DZ1;
	double * const __restrict__ R2 = solver->DZ2;
	double * const __restrict__ R3 = solver->DZ3;

	#pragma unroll
	for (int i = 0; i < NSP; i++) {
		R1[INDEX(i)] = Z1[INDEX(i)];
		R2[INDEX(i)] = Z2[INDEX(i)];
		R3[INDEX(i)] = Z3[INDEX(i)];
	}

	// TMP = Y + Z1
	WADD(Y, Z1, TMP);
	dydt(t + rkC[0] * H, pr, TMP, F, mech);
	//R[:] -= -h * rkA[:][0] * F[:]
	DAXPY3(-H * rkA[0][0], -H * rkA[1][0], -H * rkA[2][0], F, R1, R2, R3);

	// TMP = Y + Z2
	WADD(Y, Z2, TMP);
	dydt(t + rkC[1] * H, pr, TMP, F, mech);
	//R[:] -= -h * rkA[:][1] * F[:]
	DAXPY3(-H * rkA[0][1], -H * rkA[1][1], -H * rkA[2][1], F, R1, R2, R3);

	// TMP = Y + Z3
	WADD(Y, Z3, TMP);
	dydt(t + rkC[2] * H, pr, TMP, F, mech);
	//R[:] -= -h * rkA[:][2] * F[:]
	DAXPY3(-H * rkA[0][2], -H * rkA[1][2], -H * rkA[2][2], F, R1, R2, R3);
}

__device__ void dlaswp(double * const __restrict__ A,
					   int const * const __restrict__ ipiv) {
	#pragma unroll 8
	for (int i = 0; i < NSP; i++) {
		int ip = ipiv[INDEX(i)];
		if (ip != i) {
			double temp = A[INDEX(i)];
			A[INDEX(i)] = A[INDEX(ip)];
			A[INDEX(ip)] = temp;
		}
	}
}

//diag == 'n' -> nounit = true
//upper == 'u' -> upper = true
__device__ void dtrsm(bool upper, bool nounit,
					  double const * const __restrict__ A,
					  double * const __restrict__ b) {
	if (upper) {
		#pragma unroll 8
		for (int k = NSP - 1; k >= 0; --k)
		{
			if (nounit) {
				b[INDEX(k)] /= A[INDEX(k + k * NSP)];
			}
			#pragma unroll 8
			for (int i = 0; i < k; i++)
			{
				b[INDEX(i)] -= b[INDEX(k)] * A[INDEX(i + k * NSP)];
			}
		}
	}
	else{
		#pragma unroll 8
		for (int k = 0; k < NSP; k++) {
			if (fabs(b[INDEX(k)]) > 0) {
				if (nounit) {
					b[INDEX(k)] /= A[INDEX(k + k * NSP)];
				}
				#pragma unroll 8
				for (int i = k + 1; i < NSP; i++)
				{
					b[INDEX(i)] -= b[INDEX(k)] * A[INDEX(i + k * NSP)];
				}
			}
		}
	}
}

__device__ void dgetrs(double * const __restrict__ A,
					   double * const __restrict__ B,
					   int const * const __restrict__ ipiv) {
	dlaswp(B, ipiv);
	dtrsm(false, false, A, B);
	dtrsm(true, true, A, B);
}

__device__ void zlaswp(hipDoubleComplex * const __restrict__ A, int const * const __restrict__ ipiv) {
	#pragma unroll 8
	for (int i = 0; i < NSP; i++) {
		int ip = ipiv[INDEX(i)];
		if (ip != i) {
			hipDoubleComplex temp = A[INDEX(i)];
			A[INDEX(i)] = A[INDEX(ip)];
			A[INDEX(ip)] = temp;
		}
	}
}

//diag == 'n' -> nounit = true
//upper == 'u' -> upper = true
__device__ void ztrsm(bool upper, bool nounit,
					  hipDoubleComplex const * const __restrict__ A,
					  hipDoubleComplex * const __restrict__ b) {
	if (upper) {
		#pragma unroll 8
		for (int k = NSP - 1; k >= 0; --k)
		{
			if (nounit) {
				b[INDEX(k)] = hipCdiv(b[INDEX(k)], A[INDEX(k + k * NSP)]);
			}
			#pragma unroll 8
			for (int i = 0; i < k; i++)
			{
				b[INDEX(i)] = hipCsub(b[INDEX(i)], hipCmul(b[INDEX(k)], A[INDEX(i + k * NSP)]));
			}
		}
	}
	else{
		#pragma unroll 8
		for (int k = 0; k < NSP; k++) {
			if (hipCabs(b[INDEX(k)]) > 0) {
				if (nounit) {
					b[INDEX(k)] = hipCdiv(b[INDEX(k)], A[INDEX(k + k * NSP)]);
				}
				#pragma unroll 8
				for (int i = k + 1; i < NSP; i++)
				{
					b[INDEX(i)] = hipCsub(b[INDEX(i)], hipCmul(b[INDEX(k)], A[INDEX(i + k * NSP)]));
				}
			}
		}
	}
}

__device__ void zgetrs(hipDoubleComplex * const __restrict__ A,
					   hipDoubleComplex * const __restrict__ B,
					   int const * const __restrict__ ipiv) {
	zlaswp(B, ipiv);
	ztrsm(false, false, A, B);
	ztrsm(true, true, A, B);
}

__device__ void RK_Solve(const double H,
								solver_memory const * const __restrict__ solver,
								hipDoubleComplex * const __restrict__ temp) {

	double* const __restrict__ E1 = solver->E1;
	hipDoubleComplex * const __restrict__ E2 = solver->E2;
	double * const __restrict__ R1 = solver->DZ1;
	double * const __restrict__ R2 = solver->DZ2;
	double * const __restrict__ R3 = solver->DZ3;
	int* const __restrict__ ipiv1 = solver->ipiv1;
	int* const __restrict__ ipiv2 = solver->ipiv2;

	// Z = (1/h) T^(-1) A^(-1) * Z
	#pragma unroll 8
	for(int i = 0; i < NSP; i++)
	{
		double x1 = R1[INDEX(i)] / H;
		double x2 = R2[INDEX(i)] / H;
		double x3 = R3[INDEX(i)] / H;
		R1[INDEX(i)] = rkTinvAinv[0][0] * x1 + rkTinvAinv[0][1] * x2 + rkTinvAinv[0][2] * x3;
		R2[INDEX(i)] = rkTinvAinv[1][0] * x1 + rkTinvAinv[1][1] * x2 + rkTinvAinv[1][2] * x3;
		R3[INDEX(i)] = rkTinvAinv[2][0] * x1 + rkTinvAinv[2][1] * x2 + rkTinvAinv[2][2] * x3;
	}
	dgetrs(E1, R1, ipiv1);
	#pragma unroll 8
	for (int i = 0; i < NSP; ++i)
	{
		temp[INDEX(i)] = make_hipDoubleComplex(R2[INDEX(i)], R3[INDEX(i)]);
	}
	zgetrs(E2, temp, ipiv2);
	#pragma unroll 8
	for (int i = 0; i < NSP; ++i)
	{
		R2[INDEX(i)] = hipCreal(temp[INDEX(i)]);
		R3[INDEX(i)] = hipCimag(temp[INDEX(i)]);
	}

	// Z = T * Z
	#pragma unroll 8
	for (int i = 0; i < NSP; ++i) {
		double x1 = R1[INDEX(i)];
		double x2 = R2[INDEX(i)];
		double x3 = R3[INDEX(i)];
		R1[INDEX(i)] = rkT[0][0] * x1 + rkT[0][1] * x2 + rkT[0][2] * x3;
		R2[INDEX(i)] = rkT[1][0] * x1 + rkT[1][1] * x2 + rkT[1][2] * x3;
		R3[INDEX(i)] = rkT[2][0] * x1 + rkT[2][1] * x2 + rkT[2][2] * x3;
	}
}

__device__ double RK_ErrorNorm(double const * const __restrict__ scale,
							   double const * const __restrict__ DY) {
	double sum = 0;
	#pragma unroll 8
	for (int i = 0; i < NSP; ++i){
		sum += (scale[INDEX(i)] * scale[INDEX(i)] * DY[INDEX(i)] * DY[INDEX(i)]);
	}

/*
	if (sum != sum && T_ID == 0){
		for (int i = 0; i < NSP; i++){
			printf("scale[INDEX(%d)] = %0.5g; DY[INDEX(%d)] = %0.5g\n", i, scale[INDEX(i)], i, DY[INDEX(i)]);
		}
	}
*/
	return fmax(sqrt(sum / ((double)NSP)), 1e-10);
}

__device__ double RK_ErrorEstimate(const double H, const double t,
											 const double pr,
											 double const * const __restrict__ Y,
											 solver_memory const * const __restrict__ solver,
											 mechanism_memory const * const __restrict__ mech,
											 const bool FirstStep, const bool Reject) {

	double HrkE1  = rkE[1]/H;
    double HrkE2  = rkE[2]/H;
    double HrkE3  = rkE[3]/H;

	double * const __restrict__ E1 = mech->jac;
	const double * const __restrict__ F0 = mech->dy;
    double * const __restrict__ F1 = solver->work1;
    double * const __restrict__ F2 = solver->work2;
    double * const __restrict__ TMP = solver->work3;
    double const * const __restrict__ Z1 = solver->Z1;
    double const * const __restrict__ Z2 = solver->Z2;
    double const * const __restrict__ Z3 = solver->Z3;
    int const * __restrict__ ipiv1 = solver->ipiv1;
    double const * __restrict__ scale = solver->scale;

    #pragma unroll 8
    for (int i = 0; i < NSP; ++i) {
    	F2[INDEX(i)] = HrkE1 * Z1[INDEX(i)] + HrkE2 * Z2[INDEX(i)] + HrkE3 * Z3[INDEX(i)];
    }
    #pragma unroll 8
    for (int i = 0; i < NSP; ++i) {
    	TMP[INDEX(i)] = rkE[0] * F0[INDEX(i)] + F2[INDEX(i)];
    }
    dgetrs(E1, TMP, ipiv1);
    double Err = RK_ErrorNorm(scale, TMP);
    if (Err >= 1.0 && (FirstStep || Reject)) {
        #pragma unroll 8
    	for (int i = 0; i < NSP; i++) {
        	TMP[INDEX(i)] += Y[INDEX(i)];
        }
    	dydt(t, pr, TMP, F1, mech);
    	#pragma unroll 8
    	for (int i = 0; i < NSP; i++) {
        	TMP[INDEX(i)] = F1[INDEX(i)] + F2[INDEX(i)];
        }
        dgetrs(E1, TMP, ipiv1);
        Err = RK_ErrorNorm(scale, TMP);
    }
    return Err;
}

/**
 *  5th-order Radau2A implementation
 *
 */
__device__ void integrate (const double t_start,
							const double t_end,
							const double var,
							double * const __restrict__ y,
							mechanism_memory const * const __restrict__ mech,
							solver_memory const * const __restrict__ solver) {
	double Hmin = 0;
	double Hold = 0;
#ifdef Gustafsson
	double Hacc = 0;
	double ErrOld = 0;
#endif
#ifdef CONST_TIME_STEP
	double H = t_end - t_start;
#else
	double H = fmin(5e-7, t_end - t_start);
#endif
	double Hnew;
	double t = t_start;
	bool Reject = false;
	bool FirstStep = true;
	bool SkipJac = false;
	bool SkipLU = false;
	bool UseAccurateJac = false;

	double * const __restrict__ A = mech->jac;
	double * const __restrict__ sc = solver->scale;
	double * const __restrict__ y0 = solver->y0;
	double * const __restrict__ F0 = mech->dy;
	double * const __restrict__ work1 = solver->work1;
	double * const __restrict__ work2 = solver->work2;
	hipDoubleComplex * const __restrict__ work4 = solver->work4;
	double * const __restrict__ Z1 = solver->Z1;
	double * const __restrict__ Z2 = solver->Z2;
	double * const __restrict__ Z3 = solver->Z3;
	double * const __restrict__ DZ1 = solver->DZ1;
	double * const __restrict__ DZ2 = solver->DZ2;
	double * const __restrict__ DZ3 = solver->DZ3;
	double * const __restrict__ CONT = solver->CONT;
	int * const __restrict__ result = solver->result;

	scale_init(y, sc);
	safe_memcpy(y0, y);
#ifndef FORCE_ZERO
	safe_memset(F0, 0.0);
#endif
	int info = 0;
	int Nconsecutive = 0;
	int Nsteps = 0;
	double NewtonRate = pow(2.0, 1.25);
	while (t * ( 1.0 + Roundoff)  < t_end) {
		#ifdef DIVERGENCE_TEST
			integrator_steps[T_ID]++;
		#endif
		if(!Reject) {
			dydt (t, var, y, F0, mech);
		}
		if(!SkipLU) {
			//need to update Jac/LU
			if(!SkipJac) {
#ifndef FINITE_DIFFERENCE
                UseAccurateJac = false;
				if (!UseAccurateJac){
					eval_jacob (t, var, y, A, mech);
				} else {
					eval_jacob_accurate (t, var, y, A, mech);
				}
#else
				eval_jacob (t, var, y, A, mech, work1, work2);
#endif
			}
			RK_Decomp(H, A, solver, &info);
			if(info != 0) {
				Nconsecutive += 1;
				if (Nconsecutive >= 5)
				{
					result[T_ID] = EC_consecutive_steps;
					return;
				}
				H *= 0.5;
				Reject = true;
				SkipJac = true;
				SkipLU = false;
				continue;
			}
			else
			{
				Nconsecutive = 0;
			}
		}
		Nsteps += 1;
		if (Nsteps >= Max_no_steps)
		{
			result[T_ID] = EC_max_steps_exceeded;
			return;
		}
		if (0.1 * fabs(H) <= fabs(t) * Roundoff)
		{
			result[T_ID] = EC_h_plus_t_equals_h;
			if( T_ID == 0 ){
				printf("stops at t = %0.5g; target = %0.5g; H = %0.5g\n", t, t_end, H);
				for (int i = 0; i < NSP; i++){
					printf("y[INDEX(%d)] = %0.5g\n", i, y[INDEX(i)]);
				}
			}
			return;
		}
		if (FirstStep || !StartNewton) {
			safe_memset3(Z1, Z2, Z3, 0.0);
		} else {
			RK_Interpolate(H, Hold, Z1, Z2, Z3, CONT);
		}
		bool NewtonDone = false;
		double NewtonIncrementOld = 0;
		double Fac = 0.5; //Step reduction if too many iterations
		int NewtonIter = 0;
		double Theta = 0;

		//reuse previous NewtonRate
		NewtonRate = pow(fmax(NewtonRate, EPS), 0.8);

		for (; NewtonIter < NewtonMaxit; NewtonIter++) {
			RK_PrepareRHS(t, var, H, y, solver, mech, work1, work2);
			RK_Solve(H, solver, work4);
			double d1 = RK_ErrorNorm(sc, DZ1);
			double d2 = RK_ErrorNorm(sc, DZ2);
			double d3 = RK_ErrorNorm(sc, DZ3);
			double NewtonIncrement = sqrt((d1 * d1 + d2 * d2 + d3 * d3) / 3.0);

			if ( d1 != d1 && T_ID == 0 ) printf("NewtonIncrement is Nan at time = t, and H = %0.5g \n",t, H );


			Theta = ThetaMin;
			if (NewtonIter > 0)
			{
				Theta = NewtonIncrement / NewtonIncrementOld;
				if(Theta >= 0.99) //! Non-convergence of Newton: Theta too large
					break;
				else
					NewtonRate = Theta / (1.0 - Theta);
				//Predict error at the end of Newton process
				double NewtonPredictedErr = (NewtonIncrement * pow(Theta, (NewtonMaxit - NewtonIter - 1))) / (1.0 - Theta);
				if(NewtonPredictedErr >= NewtonTol) {
					//Non-convergence of Newton: predicted error too large
					double Qnewton = fmin(10.0, NewtonPredictedErr / NewtonTol);
                    Fac = 0.8 * pow(Qnewton, -1.0/((double)(NewtonMaxit-NewtonIter)));
                  	break;
				}
			}

			NewtonIncrementOld = fmax(NewtonIncrement, Roundoff);
            // Update solution
            #pragma unroll 8
            for (int i = 0; i < NSP; i++)
            {
            	Z1[INDEX(i)] -= DZ1[INDEX(i)];
            	Z2[INDEX(i)] -= DZ2[INDEX(i)];
            	Z3[INDEX(i)] -= DZ3[INDEX(i)];


            }

            NewtonDone = (NewtonRate * NewtonIncrement <= NewtonTol);

            /*
            if (T_ID == 0){
                printf("SkipJac = %d, NewtonRate = %0.5g; ThetaMin = %0.5g Theta = %0.5g at time = %0.5g\n", SkipJac, NewtonRate, ThetaMin, Theta, t);
                printf("NewtonRate*NewtonIncrement = %0.5g; NewtonTol = %0.5g; NewtonIter = %d \n" ,NewtonRate * NewtonIncrement, NewtonTol, NewtonIter);
            }
            */


            if (NewtonDone) break;
            if (NewtonIter >= NewtonMaxit)
            {
				result[T_ID] = EC_newton_max_iterations_exceeded;
				return;
			}
		}
#ifndef CONST_TIME_STEP
		if(!NewtonDone) {
			H = Fac * H;
			Reject = true;
			SkipJac = true;
			SkipLU = false;
			continue;
		}

		double Err = RK_ErrorEstimate(H, t, var, y,
						solver, mech, FirstStep, Reject);


		//!~~~> Computation of new step size Hnew
		Fac = pow(Err, (-1.0 / rkELO)) * (1.0 + 2 * NewtonMaxit) / (NewtonIter + 1.0 + 2 * NewtonMaxit);
		Fac = fmin(FacMax, fmax(FacMin, Fac));
		Hnew = Fac * H;

            if (T_ID == 0){
                printf("Fac: %0.5g; Err =%0.5g; H %0.5g Hnew %0.5g \n", Fac, Err, H, Hnew);
            }


		bool FailedIntegration = false;
		double tfail = 0.0;
		if (Err < 1.0) {
///////////////////////////////////////////////////
		for (int i = 0; i < NSP; i++){
			if ( (Z3[INDEX(i)] != Z3[INDEX(i)]) || ( (Z3[INDEX(i) ] + y[INDEX(i)]) < 0.0 ) ){
				Err = 1.0e10;
				//!~~~> Computation of new step size Hnew
				Fac = pow(Err, (-1.0 / rkELO)) * (1.0 + 2 * NewtonMaxit) / (NewtonIter + 1.0 + 2 * NewtonMaxit);
				Fac = fmin(FacMax, fmax(FacMin, Fac));
				Hnew = Fac * H;

				Reject = true;
				SkipJac = false;
				SkipLU = false;
				FailedIntegration = true;
				UseAccurateJac = true;
				if (FirstStep || Reject) {
					H = FacRej * H;
				} else {
					H = Hnew;
				}
				if( T_ID == 0 ) printf("t = %0.5g; err < 1.0 YET Z3 is Nan; reducing with Fac = %0.5g; H = %0.5g; Hnew = %0.5g \n",t,  Fac, H, Hnew);
				break;
			}
		}
		if (FailedIntegration) continue;
//////////////////////////////////////////////////////////

#ifdef Gustafsson
			if (!FirstStep) {
				double FacGus = FacSafe * (H / Hacc) * pow(Err * Err / ErrOld, -0.25);
				FacGus = fmin(FacMax, fmax(FacMin, FacGus));
				Fac = fmin(Fac, FacGus);
				Hnew = Fac * H;
                /*
            if (T_ID == 0){
                printf("Fac: %0.5g; Err =%0.5g; FacGus %0.5g, H %0.5g Hnew %0.5g \n", Fac, Err, FacGus,H, Hnew);
            }
*/

			}
			Hacc = H;
			ErrOld = fmax(1e-2, Err);
#endif
			FirstStep = false;
			Hold = H;
			t += H;
			#pragma unroll 8
			for (int i = 0; i < NSP; i++) {
				y[INDEX(i)] += Z3[INDEX(i)];
				if (y[INDEX(i)] != y[INDEX(i)]) {
					if (T_ID == 0){
						printf("IT BREAKS, with Err = %0.5g!!! at time = %0.5g, Hnew = %0.5g, H = %0.5g\n",Err, t, Hnew, H);
						for ( int j = 0; j < NSP; j++){
							printf("y_current[%d] = %0.5g; dy = %0.5g; Z1 = %0.5g; Z2 = %0.5g;  Z3 = %0.5g\n", j, y[INDEX(j)], F0[INDEX(j)], Z1[INDEX(j)], Z2[INDEX(j)], Z3[INDEX(j)] );
						}
					}

					H = FacRej * H;
					Reject  = true;
					SkipJac = false;
					SkipLU  = false;
					FirstStep = false;
					break;
				}

			}
			// Construct the solution quadratic interpolant Q(c_i) = Z_i, i=1:3
			if (StartNewton) {
				RK_Make_Interpolate(Z1, Z2, Z3, CONT);
			}
			scale(y, y0, sc);
			safe_memcpy(y0, y);
			Hnew = fmin(fmax(Hnew, Hmin), t_end - t);
			if (Reject) {
				Hnew = fmin(Hnew, H);
			}
			Reject = false;
			if (t + Hnew / Qmin - t_end >= 0.0) {
				H = t_end - t;
			} else {
				double Hratio = Hnew / H;
	            // Reuse the LU decomposition
	            SkipLU = (Theta <= ThetaMin) && (Hratio>=Qmin) && (Hratio<=Qmax);
                /*
                if (T_ID ==0 )
                {
                    printf("SkipLU = %d, H = %0.5g; Hnew = %0.5g; Fac = %0.5g\n", SkipLU, H, Hnew, Fac);
                }
                */
	            if (!SkipLU) H = Hnew;
			}
			// If convergence is fast enough, do not update Jacobian
         	SkipJac =  NewtonIter == 0 || NewtonIter == 1 || NewtonRate <= ThetaMin;
            /*
            if (T_ID == 0){
                printf("SkipJac = %d, NewtonIter = %d, NewtonRate = %0.5g; ThetaMin = %0.5g Theta = %0.5g at time = %0.5g\n", SkipJac, NewtonIter , NewtonRate, ThetaMin, Theta, t);
                printf("Reject: %d \n", Reject);
            }
            */

/////////////////////////////////////////////////////////////////////////
		// try to rescale the input when some value of the y
		// falls below the ATOL;

#ifdef RESCALE_SOLVER
	if (Nsteps % 10000 == 0){
		#pragma unroll 8
		for (int i = 0; i < NSP; i++) {
			if ( y[INDEX(i)] < RTOL ){
				mech->scale[INDEX(i)] *= y[INDEX(i)];
				mech->inv_scale[INDEX(i)] = 1.0 / mech->scale[INDEX(i)];
				y[INDEX(i)] = 1.0;
				if (T_ID == 0) {
					printf("rescaling at t= %0.5g\n", t);
				}
				Reject = false;
				SkipJac= false;
				SkipLU = false;
				FirstStep = true;
			}
		}
	}
#endif

//////////////////////////////////////////////////////////////////////////////////////////
		}


		else {
			if (FirstStep || Reject) {
				H = FacRej * H;
			} else {
				H = Hnew;
			}
			Reject = true;
			SkipJac = true;
			SkipLU = false;
		/*
			if ( T_ID == 0){
				printf("at time = %0.5g\n", t);
				for ( int i = 0; i < NSP; i++){
					printf("y_current[%d] = %0.5g\n", i, y[INDEX(i)]);
				}
			}
*/

		}
#else
		//constant time stepping
		//update y & t
		t += H;
		#pragma unroll 8
		for (int i = 0; i < NSP; i++) {
			y[INDEX(i)] += Z3[INDEX(i)];
		}
#endif
	}
	result[T_ID] = EC_success;

	if (T_ID == 0) printf("finishes at time = %0.5g; t_end = %0.5g\n", t, t_end);
}
