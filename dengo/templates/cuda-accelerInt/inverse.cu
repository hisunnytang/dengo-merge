#include "hip/hip_runtime.h"
/**
 * \file
 * \brief CUDA LU decomposition implementation
 */

#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "header.cuh"
#include "solver_props.cuh"

///////////////////////////////////////////////////////////

/**
 *  \brief getMax finds the index of the first element having maximum absolute value.
 *
 *  \param[in]          n               The size of Arr
 *  \param[in]          Arr      		The (nx1) vector to determine the maximum value of
 */
__device__
int getMax (const int n, const double * __restrict__ Arr) {

	int maxInd = 0;
	if (n == 1)
		return maxInd;

	double maxVal = fabs(Arr[INDEX(0)]);
	for (int i = 1; i < n; ++i) {
		if (fabs(Arr[INDEX(i)]) > maxVal) {
			maxInd = i;
			maxVal = fabs(Arr[INDEX(i)]);
		}
	}

	return maxInd;
}

///////////////////////////////////////////////////////////

/**
 * \brief scale multiplies a vector (with increment equal to one) by a constant val.
 * \param[in]       n           The vector size
 * \param[out]      val         The value to scale by
 * \param[out]      arrX        The vector to scale
 *
 */
__device__
void scale (const int n, const double val, double* __restrict__ arrX) {

	for (int i = 0; i < n; ++i) {
		arrX[INDEX(i)] *= val;
	}

}

///////////////////////////////////////////////////////////

/**
 * \brief interchanges two vectors arrX and arrY.
 *
 * \param[in]			n			the vector size
 * \param[in]			arrX		the first vector to swap
 * \param[in]			incX		the increment of the arrX vector
 * \param[in]			arrY		the second vector to swap
 * \param[in]			incY		the increment of the arrY vector
 */
__device__
void swap (const int n, double* __restrict__ arrX, const int incX, double* __restrict__ arrY, const int incY) {

	int ix = 0;
	int iy = 0;

	for (int i = 0; i < n; ++i) {
		double temp = arrX[INDEX(ix)];
		arrX[INDEX(ix)] = arrY[INDEX(iy)];
		arrY[INDEX(iy)] = temp;
		ix += incX;
		iy += incY;
	}

}

///////////////////////////////////////////////////////////

/**
 * \brief GERU performs the rank 1 operation \f$A := alpha * arrX * arrY **T + A\f$
          where alpha is a scalar, arrX and arrY are n element vectors, and A is a (lda x n) matrix
 *
 * \param[in]           n           The matrix/vector size
 * \param[in]           alpha       The value to scale by
 * \param[in]           arrX        arrX is an array of dimension at least n.
                                    Before entry, the incremented array arrX must contain the n
                                    element vector x.
 * \param[in]           arrY        arrY is an array of dimension at least 1 + (n - 1) * incY.
                                    Before entry, the incremented array arrY must contain the n
                                    element vector y.
 * \param[in]           incY        On entry, INCY specifies the increment for the elements of arrY. incY must not be zero.
 * \param[out]          A           A is an array of dimension (lda x n).
                                    Before entry, the leading n by n part of the array A must
                                    contain the matrix of coefficients. On exit, A is
                                    overwritten by the updated matrix.
 * \param[in]           lda         On entry, lda specifies the first dimension of A as declared
                                    in the calling (sub) program. lda must be at least
                                    max( 1, n ).
 */
__device__
void GERU (const int n, const double alpha, const double* __restrict__ arrX,
				const double* __restrict__ arrY, const int incY, double* __restrict__ A, const int lda) {

	for (int j = 0; j < n; ++j) {
    	if (fabs(arrY[INDEX(j * incY)]) > 0.0) {

			double temp = alpha * arrY[INDEX(j * incY)];

			for (int i = 0; i < n; ++i) {
				A[INDEX(i + (lda * j))] += arrX[INDEX(i)] * temp;
			}

		}
	}

}

///////////////////////////////////////////////////////////

/**
 * \brief Computes the LU factorization of a (n x n) matrix using partial pivoting with row interchanges.
          @see STRIDE
 * \param[in]       n           The matrix size
 * \param[in,out]   A           The matrix to factorize (n x n) with stride defined in solver_props.h @see STRIDE
 * \param[out]      indPivot    indPivot is an array of dimension (n).
                                The pivot indices from getHessenbergLU; for 0<=i<=n-1, row i of the
                                matrix was interchanged with row indPiv[i].
 & \param[out]      info        An information variable
 *
 * The factorization has the form:
        \f$A = P * L * U\f$
 *  where P is a permutation matrix, L is lower triangular with unit
 *  diagonal elements (lower trapezoidal if m > n), and U is upper
 *  triangular (upper trapezoidal if m < n).
 */
__device__
void getLU (const int n, double* __restrict__ A, int* __restrict__ indPivot, int* __restrict__ info) {

	for (int j = 0; j < n; ++j) {

		// find pivot and test for singularity

		int jp = j + getMax (n - j, &A[GRID_DIM * (j + (STRIDE * j))]);
		indPivot[INDEX(j)] = jp;

    	if (fabs(A[INDEX(jp + (STRIDE * j))]) > 0.0) {

			// apply interchange to columns 1:n-1
			if (jp != j)
				swap(n, &A[GRID_DIM * (j)], STRIDE, &A[GRID_DIM * (jp)], STRIDE);

			// compute elements j+1:m-1 of the jth column

			if (j < n - 1)
				scale(n - j - 1, 1.0 / A[INDEX(j + (STRIDE * j))], &A[GRID_DIM * (j + 1 + (STRIDE * j))]);

		} else if (*info == 0) {
			*info = j;
			break;
		}

		// update trailing submatrix
		if (j < n - 1)
			GERU (n - j - 1, -1.0, &A[GRID_DIM * (j + 1 + (STRIDE * j))], &A[GRID_DIM * (j + STRIDE * (j + 1))], STRIDE, &A[GRID_DIM * (j + 1 + STRIDE * (j + 1))], STRIDE);
	}
}
